#include "hip/hip_runtime.h"
#include <stdio.h>
#include "handleDataset.h"
#include <time.h>       /* time_t, time (for timestamp in second) */
#include <sys/timeb.h>  /* ftime, timeb (for timestamp in millisecond) */
#include "cuda_reduce.cu"

//#include <hipcub/hipcub.hpp>
//#include "Utilities.cuh"

using namespace std;

#define CONNECTIONS "data_small.csv"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//sqrt(sum((y-x)**2))

void sauron_eye(float *old_pk, float *new_pk, int *empty_cols, int *row_indices, int *columns,
	float *data, float *damping, int *pk_len, int *data_len, int *empty_cols_len){

	// printf("P1 damping %.8f\n",*damping);
	// printf("P1 pk_len %d\n",*pk_len);
	// printf("P1 data_len %d\n",*data_len);


	int block_number = (*pk_len + BLOCKSIZE - 1) / BLOCKSIZE;
	int uniform_blocks = (*empty_cols_len + BLOCKSIZE - 1)/BLOCKSIZE;
	int mul_blocks = (*data_len + BLOCKSIZE -1)/BLOCKSIZE;
	//printf("Block number: %d\n", block_number);
	
	float *result, *out, *out_unif, *empty_contrib, *empty_value, *weighted;
	bool *loop;
	hipMalloc(&result, sizeof(float));
	hipMallocManaged(&empty_contrib, sizeof(float));
	hipMallocManaged(&loop, sizeof(bool));
	hipMalloc(&out, sizeof(float)*block_number);
	hipMalloc(&out_unif, sizeof(float)*block_number);
	hipMallocManaged(&empty_value, sizeof(float));
	hipMalloc(&weighted, *pk_len*sizeof(float));

	
	int i = 0;

	float * tmp;

	float teleportation = DAMPING_F/ *pk_len;
	hipMemcpy(empty_value, &teleportation, sizeof(float),hipMemcpyHostToDevice);

	
	*loop = true;
	
	while (*loop){
		printf("-------------- Iteration %d ----------------\n", i);
		if (i!=0){
			tmp = old_pk;
			old_pk = new_pk;
			new_pk = tmp;
			hipMemset(new_pk, 0, *pk_len*sizeof(float));
		}


		uniform_reduction <BLOCKSIZE> <<<uniform_blocks, BLOCKSIZE, BLOCKSIZE *sizeof(float)>>> (old_pk, empty_cols, out_unif, empty_value, *empty_cols_len);
		cuda_reduction <BLOCKSIZE> <<< 1, BLOCKSIZE, BLOCKSIZE*sizeof(float)>>>(out_unif, empty_contrib, uniform_blocks);	//ok	

		pk_multiply<BLOCKSIZE> <<<mul_blocks, BLOCKSIZE>>>(data, columns, row_indices, old_pk, new_pk, *data_len, pk_len);


		*loop = false;


		sumAll<BLOCKSIZE> <<< block_number, BLOCKSIZE >>> (empty_contrib, damping, new_pk, pk_len);

		check_termination<1> <<<1, 1>>>(old_pk, new_pk, out, result, loop, pk_len, block_number);
		printf("Check termination\n");

		i++;

		hipDeviceSynchronize();
		//if (i == 1) break;
	}

	hipFree(result);
	hipFree(loop);
	hipFree(out);

}


int main(){

    int nodes_number, col_indices_number, empty_len;
	float damping;
	
	string datasetPath = CONNECTIONS;

	loadDimensions(datasetPath, nodes_number, col_indices_number, damping, empty_len);

	int *row_ptrs = (int*) malloc(col_indices_number * sizeof(int));
	int *col_indices = (int*) malloc(col_indices_number * sizeof(int));
	int *empty_cols = (int*) malloc(empty_len * sizeof(int));
	float *connections = (float*) malloc(col_indices_number * sizeof(float));

	cout << "Allocated vectors succesfully!" << endl;
	
	loadDataset(datasetPath, row_ptrs, col_indices, connections, empty_cols);

	cout << "Allocate and initialize PageRank" << endl;

	cout << "Nodes: " << nodes_number << endl;
	
	float *pr = (float*) malloc(nodes_number*sizeof(float));
	float uniform_p = 1/(float)nodes_number;
	// cout << "Uniform_p " << uniform_p << endl;
	for (int i = 0; i < nodes_number; i++){
		pr[i] = uniform_p;
	}

	cout << "Finished allocation" << endl;


	// GPU variables
	float *pk_gpu, *new_pk, *factor_gpu, *d_gpu;
	int *c_gpu, *r_gpu, *data_len, *pk_len, *empty_len_gpu, *empty_gpu;

	// Allocate device memory

	// int empty_columns = 1;
	// int empty_c[] = {2}; 

	hipMalloc(&pk_gpu, nodes_number*sizeof(float));
	hipMalloc(&new_pk, nodes_number*sizeof(float));
	hipMalloc(&factor_gpu, sizeof(float));
	hipMalloc(&c_gpu, col_indices_number*sizeof(int));
	hipMalloc(&d_gpu, col_indices_number*sizeof(float));
	hipMalloc(&r_gpu, col_indices_number*sizeof(int));
	hipMallocManaged(&pk_len, sizeof(int));
	hipMallocManaged(&data_len, sizeof(int));
	hipMallocManaged(&empty_len_gpu, sizeof(int));
	hipMalloc(&empty_gpu, empty_len*sizeof(int));

	// Populate device data from main memory

	cout << "DAMPING FROM CSV: " << damping << endl;


	hipMemcpy(pk_gpu, pr, nodes_number*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(factor_gpu, &damping, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(c_gpu, col_indices, sizeof(int)*col_indices_number, hipMemcpyHostToDevice);
	hipMemcpy(d_gpu, connections, sizeof(float)*col_indices_number, hipMemcpyHostToDevice);
	hipMemcpy(r_gpu, row_ptrs, sizeof(int)*col_indices_number, hipMemcpyHostToDevice);	
	hipMemcpy(pk_len, &nodes_number, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(data_len, &col_indices_number, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(empty_len_gpu, &empty_len, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(empty_gpu, empty_cols, sizeof(int)*empty_len, hipMemcpyHostToDevice);	

	// Get timestamp
	struct timeb timer_msec;
	long long int timestamp_start, timestamp_end; /* timestamp in millisecond. */
	if (!ftime(&timer_msec)) {
	  timestamp_start = ((long long int) timer_msec.time) * 1000ll + 
						  (long long int) timer_msec.millitm;
	}
	else {
	  timestamp_start = -1;
	}


	
	sauron_eye(pk_gpu, new_pk, empty_gpu, r_gpu, c_gpu, d_gpu, factor_gpu, pk_len, data_len, empty_len_gpu);

	gpuErrchk( hipDeviceSynchronize() );

	// Copy data back
	hipMemcpy(pr, new_pk, nodes_number*sizeof(float), hipMemcpyDeviceToHost);

	if (!ftime(&timer_msec)) {
		timestamp_end = ((long long int) timer_msec.time) * 1000ll + 
							(long long int) timer_msec.millitm;
		}
	else {
	timestamp_end = -1;
	}

	printf("--------Finished--------\n");

	cout << "Time to convergence: " << (float)(timestamp_end - timestamp_start) / 1000 << endl;

	hipFree(new_pk);
	hipFree(pk_gpu);
	hipFree(r_gpu);
	hipFree(c_gpu);
	hipFree(d_gpu);
	hipFree(factor_gpu);
	hipFree(pk_len);
	hipFree(data_len);
	hipFree(empty_gpu);
	hipFree(empty_len_gpu);

	for (int i = 0; i < 3; i++){
		cout << pr[i] << endl; 
	}

	storePagerank(pr, nodes_number, "pk_data_small.csv");
	
	return 0;
}