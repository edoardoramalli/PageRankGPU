#include <stdio.h>
#include <unistd.h>
#include "handleDataset.h"
#include <time.h>       /* time_t, time (for timestamp in second) */
#include <sys/timeb.h>  /* ftime, timeb (for timestamp in millisecond) */
#include "cuda_reduce.cu"

using namespace std;

#define DAMPING_F 0.85
#define THRESHOLD 0.000001
#define SYNCHRONIZE 1 /* Remove definition to disable "extra" deviceSynchronize calls after kernel launch */



#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
/* Check for eventual CUDA errors */
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void sauronEye(float* pkCPU, float *oldPkGPU, float *newPkGPU, int *emptyColIndices, int *rowIndicesGPU, int *columnIndicesGPU,
	float *matrixDataGPU, float *dampingMatrixFactorGPU, int *pkLenGPU, int *matrixDataLenGPU,
	 int *emptyColIndicesLenGPU, int dampingFactor, float precisionThreshold){

	/* Concealed within his fortress, the lord of Mordor sees all.
	His gaze pierces cloud, shadow, earth, and flesh.
	You know of what I speak, Gandalf: a great Eye, lidless, wreathed in flame. */

	int blockNumber = (*pkLenGPU + BLOCKSIZE - 1) / BLOCKSIZE;
	int uniformReductionBlocks = (*emptyColIndicesLenGPU + BLOCKSIZE - 1)/BLOCKSIZE;
	int mulBlocks = (*matrixDataLenGPU + BLOCKSIZE -1)/BLOCKSIZE;
	
	float *result, *out, *emptyColumnsContrib, *emptyColumnValue, *thresholdGPU;
	bool *loop;

	hipMalloc(&thresholdGPU, sizeof(float));
	hipMalloc(&result, sizeof(float));
	hipMalloc(&emptyColumnsContrib, sizeof(float));
	hipMallocManaged(&loop, sizeof(bool));
	hipMalloc(&out, sizeof(float)*blockNumber);
	hipMalloc(&emptyColumnValue, sizeof(float));

	
	int i = 0;

	float * tmp;

	float teleportation = dampingFactor/ *pkLenGPU;
	hipMemcpy(emptyColumnValue, &teleportation, sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(thresholdGPU, &precisionThreshold, sizeof(float), hipMemcpyHostToDevice);

	
	*loop = true;

	// Get timestamp
	struct timeb timerMsec;
	long long int timestampStart, timestampEnd; /* timestamp in millisecond. */
	
	while (*loop){
		*loop = false;

		if (i!=0){
			// Swap pointers to avoid allocating new memory
			tmp = oldPkGPU;
			oldPkGPU = newPkGPU;
			newPkGPU = tmp;

			hipMemset(newPkGPU, 0, *pkLenGPU*sizeof(float));	/* Set new pagerank to 0 */
		}
		else{
			// Get starting timestamp
			if (!ftime(&timerMsec)) {
				timestampStart = ((long long int) timerMsec.time) * 1000ll + 
									(long long int) timerMsec.millitm;
			}
			else {
				timestampStart = -1;
			}
		}

		uniformReduction <BLOCKSIZE> <<<uniformReductionBlocks, BLOCKSIZE, BLOCKSIZE *sizeof(float)>>> (oldPkGPU, emptyColIndices, out, emptyColumnValue, *emptyColIndicesLenGPU);
		cudaReduction <BLOCKSIZE> <<< 1, BLOCKSIZE, BLOCKSIZE*sizeof(float)>>>(out, emptyColumnsContrib, uniformReductionBlocks);		
		#ifdef SYNCHRONIZE
		hipDeviceSynchronize();
		#endif

		pkMultiply<BLOCKSIZE> <<<mulBlocks, BLOCKSIZE>>>(matrixDataGPU, columnIndicesGPU, rowIndicesGPU, oldPkGPU, newPkGPU, *matrixDataLenGPU, pkLenGPU);
		#ifdef SYNCHRONIZE
		hipDeviceSynchronize();
		#endif

		sumAll<BLOCKSIZE> <<< blockNumber, BLOCKSIZE >>> (emptyColumnsContrib, dampingMatrixFactorGPU, newPkGPU, pkLenGPU);

		checkTermination<1> <<<1, 1>>>(oldPkGPU, newPkGPU, out, result, loop, pkLenGPU, blockNumber, thresholdGPU);
		//printf("Check termination\n");

		i++;

		hipDeviceSynchronize();
	}

	// Copy matrixDataGPU back
	gpuErrchk(hipMemcpy(pkCPU, newPkGPU, *pkLenGPU * sizeof(float), hipMemcpyDeviceToHost));

	// Get ending timestamp
	if (!ftime(&timerMsec)) {
		timestampEnd = ((long long int) timerMsec.time) * 1000ll + 
							(long long int) timerMsec.millitm;
		}
	else {
	timestampEnd = -1;
	}

	cout << endl;
	cout << "Completed Convergence in " << i << " iterations" << endl;

	cout << "Time to convergence: " << (float)(timestampEnd - timestampStart) / 1000 << endl;

	// Free allocated GPU resources

	hipFree(thresholdGPU);
	hipFree(result);
	hipFree(emptyColumnsContrib);
	hipFree(loop);
	hipFree(out);
	hipFree(emptyColumnValue);
}


int main(int argc, char *argv[]){

	int verticesNumber, colIndicesLen, emptyLen;
	float dampingMatrix;
	float dampingFactor = DAMPING_F;
	float precisionThreshold = THRESHOLD;

	string inputPath = "";
	string outputPath = "";

	int opt;
	while((opt = getopt(argc, argv, "i:o:sfd:t:"))!= EOF){
		switch (opt){
			case 'i':
				inputPath = optarg;
				outputPath =  "pk_" + inputPath; 
				break;
			case 'o':
				outputPath = optarg;
				break;
			case 's':
				inputPath = "data_small.csv";
				outputPath = "pk_data_small.csv";
				break;
			case 'f':
				inputPath = "data_full.csv";
				outputPath = "pk_data_full.csv";
				break;
			case 'd':
				dampingFactor = stof(optarg);
				break;
			case 't':
				precisionThreshold = stof(optarg);
				break;
			default:
				cout << "Invalid parameter " << opt << endl;
				exit(-1);
		}
	}


	if (inputPath == ""){
		cout << "Empty input path!" << endl;
		exit(-1);
	}

	if (outputPath == ""){
		cout << "Empty output path!" << endl;
		exit(-1);
	}

	if (precisionThreshold >= 1 | precisionThreshold < 0){
		cout << "Precision too coarse! Input a precision < 1" << endl;
		exit(-1);
	}

	if (dampingFactor >= 1 | dampingFactor < 0){
		cout << "Damping too big! Input a dampingMatrixFactorGPU between 0 and 1" << endl;
		exit(-1);
	}

	cout << "Input dataset: " << inputPath << endl;
	cout << "Output file: " << outputPath << endl;
	cout << "Damping factor: " << dampingFactor << endl;
	cout << "Precision threshold: " << precisionThreshold << endl << endl;


	/*-----------------------------------------------------------------------*/

	loadDimensions(inputPath, verticesNumber, colIndicesLen, dampingMatrix, emptyLen);

	cout << "Nodes: " << verticesNumber << endl;
	
	int *rowIndices = (int*) malloc(colIndicesLen * sizeof(int));
	int *colIndices = (int*) malloc(colIndicesLen * sizeof(int));
	int *emptyColIndices = (int*) malloc(emptyLen * sizeof(int));
	float *matrixData = (float*) malloc(colIndicesLen * sizeof(float));

	cout << "Allocated matrixDataGPU vectors succesfully!" << endl;
	
	loadDataset(inputPath, rowIndices, colIndices, matrixData, emptyColIndices);

	cout << "Allocate and initialize PageRank" << endl;

	
	float *pkCPU = (float*) malloc(verticesNumber*sizeof(float));
	float pkInit = 1/(float)verticesNumber;

	for (int i = 0; i < verticesNumber; i++){
		pkCPU[i] = pkInit;
	}

	cout << "Finished allocation" << endl;


	// GPU variables
	float *pkGPU, *newPkGPU, *dampingMatrixFactorGPU, *matrixDataGPU;
	int *columnIndicesGPU, *rowIndicesGPU, *matrixDataLenGPU, *pkLenGPU, *emptyColIndicesLenGPU, *emptyIndicesGPU;

	// Allocate device memory

	hipMalloc(&pkGPU, verticesNumber*sizeof(float));
	hipMalloc(&newPkGPU, verticesNumber*sizeof(float));
	hipMalloc(&dampingMatrixFactorGPU, sizeof(float));
	hipMalloc(&columnIndicesGPU, colIndicesLen*sizeof(int));
	hipMalloc(&matrixDataGPU, colIndicesLen*sizeof(float));
	hipMalloc(&rowIndicesGPU, colIndicesLen*sizeof(int));
	hipMallocManaged(&pkLenGPU, sizeof(int));
	hipMallocManaged(&matrixDataLenGPU, sizeof(int));
	hipMallocManaged(&emptyColIndicesLenGPU, sizeof(int));
	hipMalloc(&emptyIndicesGPU, emptyLen*sizeof(int));

	// Populate device matrixDataGPU from main memory

	hipMemcpy(pkGPU, pkCPU, verticesNumber*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dampingMatrixFactorGPU, &dampingMatrix, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(columnIndicesGPU, colIndices, sizeof(int)*colIndicesLen, hipMemcpyHostToDevice);
	hipMemcpy(matrixDataGPU, matrixData, sizeof(float)*colIndicesLen, hipMemcpyHostToDevice);
	hipMemcpy(rowIndicesGPU, rowIndices, sizeof(int)*colIndicesLen, hipMemcpyHostToDevice);	
	hipMemcpy(pkLenGPU, &verticesNumber, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(matrixDataLenGPU, &colIndicesLen, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(emptyColIndicesLenGPU, &emptyLen, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(emptyIndicesGPU, emptyColIndices, sizeof(int)*emptyLen, hipMemcpyHostToDevice);	

	// Start algorithm iteration
	
	sauronEye(pkCPU, pkGPU, newPkGPU, emptyIndicesGPU, rowIndicesGPU, columnIndicesGPU,
		 matrixDataGPU, dampingMatrixFactorGPU, pkLenGPU, matrixDataLenGPU,
		  emptyColIndicesLenGPU, dampingFactor, precisionThreshold);

	hipFree(newPkGPU);
	hipFree(pkGPU);
	hipFree(rowIndicesGPU);
	hipFree(columnIndicesGPU);
	hipFree(matrixDataGPU);
	hipFree(dampingMatrixFactorGPU);
	hipFree(pkLenGPU);
	hipFree(matrixDataLenGPU);
	hipFree(emptyIndicesGPU);
	hipFree(emptyColIndicesLenGPU);

	cout << endl;
	
	cout << "Writing output file..." << endl;

	storePagerank(pkCPU, verticesNumber, outputPath);

	cout << "Done!" << endl;
	
	return 0;
}