#include "hip/hip_runtime.h"

#include <stdio.h>

#define BLOCKSIZE 32
#define PRECISION 1000000
#define THRESHOLD 0.000001
#define DAMPING_F 0.85


template <unsigned int blockSize> __device__ void warpReduce(volatile float *sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

__device__ inline void ouratomicAdd(float* address, float value){

  float old = value;  
  float new_old;

  do{
	new_old = atomicExch(address, 0.0f);
	new_old += old;
  }
  while ((old = atomicExch(address, new_old))!=0.0f);
}

template <unsigned int blockSize> __global__ void pk_multiply(float* data, int* columns, int* rows, float* old_pk, float* new_pk, unsigned int len, int *pk_len){
	size_t tid = threadIdx.x,    
	i = blockIdx.x * blockSize + tid;
	
	if(i < len){
		float sum = data[i] * old_pk[columns[i]];
		//new_pk[rows[i]] += sum;
		ouratomicAdd(&new_pk[rows[i]], sum);
		//if (rows[i] == 5) printf("cacca %.10f\n", sum);
		
	}
}

template <unsigned int blockSize> __global__ void sumAll(float *empty_contrib, float *damping_matrix, float *new_pk, int *pk_len){
	size_t tid = threadIdx.x,    
	i = blockIdx.x * blockSize + tid;
	
	if(i < *pk_len){
		new_pk[i] += *empty_contrib + *damping_matrix;
	}
}

template <unsigned int blockSize> __global__ void cuda_reduction(float *array_in, float *reduct, size_t array_len) {
	/*Parallel block reduction*/

    extern volatile __shared__ float sdata[];
    
    size_t tid = threadIdx.x,
    gridSize = blockSize * gridDim.x,
    
    i = blockIdx.x * blockSize + tid;
    
    sdata[tid] = 0;
    
    while (i < array_len) {
        sdata[tid] += array_in[i];
        i += gridSize;
    }
    __syncthreads();
    if (blockSize >= 512) {
        if (tid < 256) sdata[tid] += sdata[tid + 256];
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (tid < 128) sdata[tid] += sdata[tid + 128];
        __syncthreads();
    }
    if (blockSize >= 128) {
        if (tid < 64) sdata[tid] += sdata[tid + 64];
        __syncthreads();
    }
    if (tid < 32) {
		warpReduce<blockSize>(sdata, tid);
	}
	
    if (tid == 0){
		reduct[blockIdx.x] = sdata[0];
	} 
}

template <unsigned int blockSize> __global__ void uniform_reduction(float *old_pk, int *empty_cols,float *reduct, float *factor, size_t array_len) {
	/* Calculate contribution from empty columns to each line:
	sum pagerank at index equal to empty column index in T',
	then multiply by the teleportation probability */

    extern volatile __shared__ float sdata[];
    
    size_t tid = threadIdx.x,
    gridSize = blockSize * gridDim.x,
    
    i = blockIdx.x * blockSize + tid;
    
	sdata[tid] = 0;
    
    while (i < array_len) {
        sdata[tid] += old_pk[empty_cols[i]];
        i += gridSize;
    }
    __syncthreads();
    if (blockSize >= 512) {
        if (tid < 256) sdata[tid] += sdata[tid + 256];
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (tid < 128) sdata[tid] += sdata[tid + 128];
        __syncthreads();
    }
    if (blockSize >= 128) {
        if (tid < 64) sdata[tid] += sdata[tid + 64];
        __syncthreads();
    }
    if (tid < 32) {
		warpReduce<blockSize>(sdata, tid);
	}
	
    if (tid == 0){
		reduct[blockIdx.x] = (*factor)*sdata[0];
	} 
}

template <unsigned int blockSize> __global__ void weighted_sum_partial(float *pagerank_in, float *reduct,
	int *column, float *mat_data, size_t row_len, size_t pk_len){
	/* Perform first step of pagerank row by column product
	by multiplying T' row by pagerank elements, only for non null T' elements.
	T is input as a CSR matrix (3 arrays: row pointers, columns, data)
	*/

	extern volatile __shared__ float sdata[];
	size_t  tid = threadIdx.x, gridSize = blockSize *gridDim.x, i = blockIdx.x * blockSize + tid;
	sdata[tid] = 0;
	while (i < row_len) {
		sdata[tid] += mat_data[i]*pagerank_in[column[i]];
		i += gridSize;
	}
	__syncthreads();
	if (blockSize >= 512) {
		if (tid < 256) sdata[tid] += sdata[tid + 256];
		__syncthreads();
	}
	if (blockSize >= 256) {
		if (tid < 128) sdata[tid] += sdata[tid + 128];
		__syncthreads();
	}
	if (blockSize >= 128) {
		if (tid <  64) sdata[tid] += sdata[tid + 64];
		__syncthreads();
	}
	if (tid < 32) {
		warpReduce<blockSize>(sdata, tid);
	}
	if (tid == 0) reduct[blockIdx.x] = sdata[0];
}


template <unsigned int blockSize> __global__ void termination_reduction(float *new_pk, float *old_pk, float *reduct, size_t array_len) {
    extern volatile __shared__ float sdata[];
    
    size_t tid = threadIdx.x,
    gridSize = blockSize * gridDim.x,
    
    i = blockIdx.x * blockSize + tid;
    
    sdata[tid] = 0;
    
    while (i < array_len) {
		float diff = new_pk[i] - old_pk[i];
        sdata[tid] += diff*diff;
        i += gridSize;
    }
    __syncthreads();
    if (blockSize >= 512) {
        if (tid < 256) sdata[tid] += sdata[tid + 256];
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (tid < 128) sdata[tid] += sdata[tid + 128];
        __syncthreads();
    }
    if (blockSize >= 128) {
        if (tid < 64) sdata[tid] += sdata[tid + 64];
        __syncthreads();
    }
    if (tid < 32) {
		warpReduce<blockSize>(sdata, tid);
	}
	
    if (tid == 0){
		reduct[blockIdx.x] = sdata[0];
	} 
}


template <unsigned int blockSize> __global__ void check_termination(float *old_pk, float *new_pk, float* out, float* result, bool *loop, 
	int *pk_len, size_t out_len){
	int block_number = (*pk_len + BLOCKSIZE - 1) / BLOCKSIZE;

	termination_reduction <BLOCKSIZE> <<<block_number, BLOCKSIZE, BLOCKSIZE*sizeof(float)>>> (new_pk, old_pk, out, *pk_len);
	cuda_reduction <BLOCKSIZE> <<<1, BLOCKSIZE, BLOCKSIZE*sizeof(float) >>> (out, result, out_len);
	hipDeviceSynchronize();
	float error;
	error = sqrtf(*result);
	printf("Error  %.10f\n", error);
	if (error > THRESHOLD) {
		*loop = true;
	}

}